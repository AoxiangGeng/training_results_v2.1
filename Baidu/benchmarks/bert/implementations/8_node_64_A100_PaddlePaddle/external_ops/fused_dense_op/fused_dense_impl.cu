// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
// Copyright Apex Library Authros. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fused_dense_cuda.h"

// FP64 Wrapper around cublas GEMMEx
// TODO(limin): in fact, alpha and beta are double type.
hipblasStatus_t gemm_bias(hipblasHandle_t handle,
                         hipblasOperation_t transa,
                         hipblasOperation_t transb,
                         int m,
                         int n,
                         int k,
                         const float* alpha,
                         const double* A,
                         int lda,
                         const double* B,
                         int ldb,
                         const float* beta,
                         double* C,
                         int ldc) {
  return hipblasGemmEx(handle,
                      transa,
                      transb,
                      m,
                      n,
                      k,
                      alpha,
                      A,
                      HIP_R_64F,
                      lda,
                      B,
                      HIP_R_64F,
                      ldb,
                      beta,
                      C,
                      HIP_R_64F,
                      ldc,
                      HIP_R_64F,
                      HIPBLAS_GEMM_DEFAULT);
}

// FP32 Wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(hipblasHandle_t handle,
                         hipblasOperation_t transa,
                         hipblasOperation_t transb,
                         int m,
                         int n,
                         int k,
                         const float* alpha,
                         const float* A,
                         int lda,
                         const float* B,
                         int ldb,
                         const float* beta,
                         float* C,
                         int ldc) {
  return hipblasGemmEx(handle,
                      transa,
                      transb,
                      m,
                      n,
                      k,
                      alpha,
                      A,
                      HIP_R_32F,
                      lda,
                      B,
                      HIP_R_32F,
                      ldb,
                      beta,
                      C,
                      HIP_R_32F,
                      ldc,
                      HIP_R_32F,
                      HIPBLAS_GEMM_DEFAULT);
}

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(hipblasHandle_t handle,
                         hipblasOperation_t transa,
                         hipblasOperation_t transb,
                         int m,
                         int n,
                         int k,
                         const float* alpha,
                         const paddle::float16* A,
                         int lda,
                         const paddle::float16* B,
                         int ldb,
                         const float* beta,
                         paddle::float16* C,
                         int ldc) {
  return hipblasGemmEx(handle,
                      transa,
                      transb,
                      m,
                      n,
                      k,
                      alpha,
                      A,
                      HIP_R_16F,
                      lda,
                      B,
                      HIP_R_16F,
                      ldb,
                      beta,
                      C,
                      HIP_R_16F,
                      ldc,
                      HIP_R_32F,
                      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600

// float64
template <>
hipblasStatus_t cublaslt_matmul_desc_init<double>(
    hipblasLtMatmulDescOpaque_t* operationDesc) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
  status =
      cublasLtMatmulDescInit(operationDesc, HIPBLAS_COMPUTE_64F, HIP_R_64F);
  return status;
}

template <>
hipblasStatus_t set_cublaslt_matrix_layout_init<float>(
    hipblasLtMatrixLayoutOpaque_t* Adesc,
    hipblasLtMatrixLayoutOpaque_t* Bdesc,
    hipblasLtMatrixLayoutOpaque_t* Cdesc,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    int lda,
    int ldb,
    int ldc) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
  status = cublasLtMatrixLayoutInit(Adesc,
                                    HIP_R_32F,
                                    transa == HIPBLAS_OP_N ? m : k,
                                    transa == HIPBLAS_OP_N ? k : m,
                                    lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(Bdesc,
                                    HIP_R_32F,
                                    transb == HIPBLAS_OP_N ? k : n,
                                    transb == HIPBLAS_OP_N ? n : k,
                                    ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(Cdesc, HIP_R_32F, m, n, ldc);
CLEANUP:
  return status;
}

template <>
hipblasStatus_t set_cublaslt_matrix_layout_init<double>(
    hipblasLtMatrixLayoutOpaque_t* Adesc,
    hipblasLtMatrixLayoutOpaque_t* Bdesc,
    hipblasLtMatrixLayoutOpaque_t* Cdesc,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    int lda,
    int ldb,
    int ldc) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
  status = cublasLtMatrixLayoutInit(Adesc,
                                    HIP_R_64F,
                                    transa == HIPBLAS_OP_N ? m : k,
                                    transa == HIPBLAS_OP_N ? k : m,
                                    lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(Bdesc,
                                    HIP_R_64F,
                                    transb == HIPBLAS_OP_N ? k : n,
                                    transb == HIPBLAS_OP_N ? n : k,
                                    ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(Cdesc, HIP_R_64F, m, n, ldc);

CLEANUP:
  return status;
}


int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    const phi::dtype::float16* A,
    int lda,
    const phi::dtype::float16* B,
    int ldb,
    const float *beta,
    phi::dtype::float16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* gelu_in,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_GELU_AUX;

  status = cublasLtMatmulDescInit(&operationDesc,
                                  HIPBLAS_COMPUTE_32F,
                                  HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_TRANSA,
                  &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_TRANSB,
                  &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                  &gelu_in, sizeof(gelu_in));
  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                  &ldc, sizeof(ldc));

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                    HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                    &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_GELU_AUX_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_EPILOGUE,
                  &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k,
                  transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n,
                  transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}


int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    const double* A,
    int lda,
    const double* B,
    int ldb,
    const float *beta,
    double* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void *gelu_in,
    const void* bias) {
  return 1;
}


int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    const float *A,
    int lda,
    const float *B,
    int ldb,
    const float *beta,
    float *C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* gelu_in,
    const void* bias) {
  return 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    const phi::dtype::float16* A,
    int lda,
    const phi::dtype::float16* B,
    int ldb,
    const float *beta,
    phi::dtype::float16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void *gelu_in,
    const void *bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

  status = cublasLtMatmulDescInit(&operationDesc,
                                  HIPBLAS_COMPUTE_32F,
                                  HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_TRANSA,
                  &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_TRANSB,
                  &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                  &bgrad, sizeof(bgrad));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                  &gelu_in, sizeof(gelu_in));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                  &ldc, sizeof(ldc));

  status = hipblasLtMatmulDescSetAttribute(&operationDesc,
                  HIPBLASLT_MATMUL_DESC_EPILOGUE,
                  &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k,
                  transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n,
                  transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    const double *A,
    int lda,
    const double *B,
    int ldb,
    const float *beta,
    double *C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void *gelu_in,
    const void *bgrad) {
    return 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    const float *A,
    int lda,
    const float *B,
    int ldb,
    const float *beta,
    float *C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void *gelu_in,
    const void *bgrad) {
    return 1;
}

#endif

